
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/batchnormp_cuda.cu"
#else

#define DeviceTensor3 THCDeviceTensor<real, 3>
#define DeviceTensor1 THCDeviceTensor<real, 1>

template <int Dim>
static THCDeviceTensor<real, Dim> devicetensor(THCState *state, THCTensor *t) {
  if (!t) {
    return THCDeviceTensor<real, Dim>();
  }

  int inDim = THCTensor_(nDimension)(state, t);
  if (inDim == Dim) {
    return toDeviceTensor<real, Dim>(state, t);
  }

  // View in which the last dimensions are collapsed or expanded as needed
  THAssert(THCTensor_(isContiguous)(state, t));
  int size[Dim];
  for (int i = 0; i < Dim || i < inDim; ++i) {
    if (i < Dim && i < inDim) {
      size[i] = t->size[i];
    } else if (i < Dim) {
      size[i] = 1;
    } else {
      size[Dim - 1] *= t->size[i];
    }
  }
  return THCDeviceTensor<real, Dim>(THCTensor_(data)(state, t), size);
}

extern "C" void THNN_(BatchNormalization_updateOutputhaha)(
  THCState *state, THCTensor *input_, THCTensor *output_,
  THCTensor *weight_, THCTensor *bias_, THCTensor *runningMean_,
  THCTensor *runningVar_, THCTensor *saveMean_, THCTensor *saveStd_,
  int train, double momentum, double eps);

extern "C" void THNN_(BatchNormalization_mean)(
  THCState *state, THCTensor *input_, THCTensor *saveMean_);

extern "C" void THNN_(BatchNormalization_var)(
  THCState *state, THCTensor *input_, THCTensor *saveMean_,
  THCTensor *saveVar_);


void THNN_(BatchNormalization_mean)(
  THCState *state, THCTensor *input_, THCTensor *saveMean_) {
  DeviceTensor3 input = devicetensor<3>(state, input_);
  DeviceTensor1 saveMean = devicetensor<1>(state, saveMean_);

  cudaStream_t s = THCState_getCurrentStream(state);
  cudaDeviceProp *prop = THCState_getCurrentDeviceProperties(state);

  dim3 blocks(input.getSize(1));
  dim3 threads(getNumThreads(input.getSize(2)));
  BatchNormalizationMean_kernel<real, accreal, DeviceTensor1, DeviceTensor3> <<<blocks, threads, 0, s>>>(
  input, saveMean);
  THCudaCheck(cudaGetLastError());
}

void THNN_(BatchNormalization_var)(
  THCState *state, THCTensor *input_, THCTensor *saveMean_, THCTensor *saveVar_) {
  DeviceTensor3 input = devicetensor<3>(state, input_);
  DeviceTensor1 saveMean = devicetensor<1>(state, saveMean_);
  DeviceTensor1 saveVar = devicetensor<1>(state, saveVar_);

  cudaStream_t s = THCState_getCurrentStream(state);
  cudaDeviceProp *prop = THCState_getCurrentDeviceProperties(state);

  dim3 blocks(input.getSize(1));
  dim3 threads(getNumThreads(input.getSize(2)));
  BatchNormalizationVar_kernel<real, accreal, DeviceTensor1, DeviceTensor3> <<<blocks, threads, 0, s>>>(
  input, saveMean, saveVar);
  THCudaCheck(cudaGetLastError());
}

void THNN_(BatchNormalization_updateOutputhaha)(
  THCState *state, THCTensor *input_, THCTensor *output_,
  THCTensor *weight_, THCTensor *bias_, THCTensor *runningMean_,
  THCTensor *runningVar_, THCTensor *saveMean_, THCTensor *saveStd_,
  int train, double momentum, double eps) {

  THCTensor_(resizeAs)(state, output_, input_);
  DeviceTensor3 input = devicetensor<3>(state, input_);
  DeviceTensor3 output = devicetensor<3>(state, output_);
  DeviceTensor1 weight = devicetensor<1>(state, weight_);
  DeviceTensor1 bias = devicetensor<1>(state, bias_);
  DeviceTensor1 runningMean = devicetensor<1>(state, runningMean_);
  DeviceTensor1 runningVar = devicetensor<1>(state, runningVar_);
  DeviceTensor1 saveMean = devicetensor<1>(state, saveMean_);
  DeviceTensor1 saveStd = devicetensor<1>(state, saveStd_);

  cudaStream_t s = THCState_getCurrentStream(state);
  cudaDeviceProp *prop = THCState_getCurrentDeviceProperties(state);

  if (!train) {
    dim3 blocks(input.getSize(1));
    dim3 threads(getNumThreads(input.getSize(2)));
    BatchNormalizationUpdateOutputInference_kernel<real, accreal, DeviceTensor1, DeviceTensor3> <<<blocks, threads, 0, s>>>(
      input, output, runningMean, runningVar, weight, bias, eps);
  } else {
    dim3 blocks(input.getSize(1));
    dim3 threads(getNumThreads(input.getSize(2)));
    BatchNormalizationUpdateOutput_kernelhaha<real, accreal, DeviceTensor1, DeviceTensor3> <<<blocks, threads, 0, s>>>(
      input, output, weight, bias, eps, momentum, runningMean, runningVar,
      saveMean, saveStd);
  }
  THCudaCheck(cudaGetLastError());
}

extern "C" void THNN_(BatchNormalization_backwardhaha)(
  THCState *state, THCTensor *input_, THCTensor *gradOutput_,
  THCTensor *gradOutputMean_, THCTensor *dotP,
  THCTensor *gradInput_, THCTensor *gradWeight_, THCTensor *gradBias_,
  THCTensor *weight_, THCTensor *runningMean_, THCTensor *runningVar_,
  THCTensor *saveMean_, THCTensor *saveStd_, int train, double scale, double eps);


extern "C" void THNN_(BatchNormalization_mean_grad)(
  THCState *state, THCTensor *input_, THCTensor *gradOutput_,
  THCTensor *runningMean_, THCTensor *saveMean_,
  THCTensor *gradOutputMean_, THCTensor *dotP_, int train);


void THNN_(BatchNormalization_mean_grad)(
  THCState *state, THCTensor *input_, THCTensor *gradOutput_,
  THCTensor *runningMean_, THCTensor *saveMean_,
  THCTensor *gradOutputMean_, THCTensor *dotP_, int train) {

  THCUNN_check_shape(state, input_, gradOutput_);
  DeviceTensor3 input = devicetensor<3>(state, input_);
  DeviceTensor3 gradOutput = devicetensor<3>(state, gradOutput_);
  DeviceTensor1 gradOutputMean = devicetensor<1>(state, gradOutputMean_);
  DeviceTensor1 dotP = devicetensor<1>(state, dotP_);

  DeviceTensor1 runningMean = devicetensor<1>(state, runningMean_);
  DeviceTensor1 saveMean = devicetensor<1>(state, saveMean_);

  cudaStream_t s = THCState_getCurrentStream(state);

  dim3 blocks(gradOutput.getSize(1));
  dim3 threads(getNumThreads(gradOutput.getSize(2)));
  BatchNormalizationMeanGrad_kernel<real,  accreal,  DeviceTensor1, DeviceTensor3> <<<blocks, threads, 0, s>>>(
    input, gradOutput, runningMean, saveMean, gradOutputMean, dotP, train);
  THCudaCheck(cudaGetLastError());
}


void THNN_(BatchNormalization_backwardhaha)(
  THCState *state, THCTensor *input_, THCTensor *gradOutput_,
  THCTensor *gradOutputMean_, THCTensor *dotP_,
  THCTensor *gradInput_, THCTensor *gradWeight_, THCTensor *gradBias_,
  THCTensor *weight_, THCTensor *runningMean_, THCTensor *runningVar_,
  THCTensor *saveMean_, THCTensor *saveStd_, int train, double scale, double eps) {

  THCUNN_check_shape(state, input_, gradOutput_);
  DeviceTensor3 input = devicetensor<3>(state, input_);
  DeviceTensor3 gradOutput = devicetensor<3>(state, gradOutput_);
  DeviceTensor1 gradOutputMean = devicetensor<1>(state, gradOutputMean_);
  DeviceTensor1 dotP = devicetensor<1>(state, dotP_);
  DeviceTensor3 gradInput = devicetensor<3>(state, gradInput_);
  DeviceTensor1 gradWeight = devicetensor<1>(state, gradWeight_);
  DeviceTensor1 gradBias = devicetensor<1>(state, gradBias_);
  DeviceTensor1 weight = devicetensor<1>(state, weight_);
  DeviceTensor1 runningMean = devicetensor<1>(state, runningMean_);
  DeviceTensor1 runningVar = devicetensor<1>(state, runningVar_);
  DeviceTensor1 saveMean = devicetensor<1>(state, saveMean_);
  DeviceTensor1 saveStd = devicetensor<1>(state, saveStd_);

  cudaStream_t s = THCState_getCurrentStream(state);

  dim3 blocks(gradOutput.getSize(1));
  dim3 threads(getNumThreads(gradOutput.getSize(2)));
  BatchNormalizationBackward_kernel<real,  accreal,  DeviceTensor1, DeviceTensor3> <<<blocks, threads, 0, s>>>(
    input, gradOutput, gradOutputMean, dotP, gradInput, gradWeight, gradBias, weight, runningMean, runningVar,
    saveMean, saveStd, train, scale, eps);
  THCudaCheck(cudaGetLastError());
}

#undef DeviceTensor3
#undef DeviceTensor1

#endif
